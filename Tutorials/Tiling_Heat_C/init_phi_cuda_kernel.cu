
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void init_phi_kernel(double *fab,
                                const int lo1, const int lo2, const int lo3,
                                const int hi1, const int hi2, const int hi3,
                                const double problo1, const double problo2, const double problo3,
                                const double probhi1, const double probhi2, const double probhi3,
                                const int jStride, const int kStride,
                                const int Nghost,
                                const double dx1, const double dx2, const double dx3) {

  int i, j, k, ijk_fab;

  double x, y, z, r2;

  // Convert CUDA thread indices into indicies of the FAB that each thread will
  // modify. Isn't this beautiful??

  i = lo1 + (blockIdx.x * blockDim.x) + threadIdx.x;
  j = lo2 + (blockIdx.y * blockDim.y) + threadIdx.y;
  k = lo3 + (blockIdx.z * blockDim.z) + threadIdx.z;

  // WARNING: for now this assumes that the thread block and the FAB have
  // identical dimensions!!

  x = problo1 + (double(i)+0.5) * dx1;
  y = problo2 + (double(j)+0.5) * dx2;
  z = problo3 + (double(k)+0.5) * dx3;

  r2 = ((x-0.25)*(x-0.25) + (y-0.25)*(y-0.25) + (z-0.25)*(z-0.25)) * 100.0;

//  ijk_fab = (i+Nghost) + (j+Nghost)*jStride + (k+Nghost)*kStride;
  ijk_fab = i + j*jStride + k*kStride;

  fab[ijk_fab] = 1.0 + std::exp(-r2);

}
