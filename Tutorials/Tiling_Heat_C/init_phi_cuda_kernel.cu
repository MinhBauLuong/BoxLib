
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void init_phi_kernel(double *fab,
                                const int lo1, const int lo2, const int lo3,
                                const int hi1, const int hi2, const int hi3,
                                const double problo1, const double problo2, const double problo3,
                                const double probhi1, const double probhi2, const double probhi3,
                                const int jStride, const int kStride,
                                const int Nghost,
                                const double dx1, const double dx2, const double dx3) {

  int i, j, k, ijk;

  double x, y, z, r2;

  for (k = lo3; k <= hi3; ++k) {
    z = problo3 + (double(k)+0.5) * dx3;
    for (j = lo2; j <= hi2; ++j) {
      y = problo2 + (double(j)+0.5) * dx2;
      for (i = lo1; i <= hi1; ++i) {
        x = problo1 + (double(i)+0.5) * dx1;

        r2 = ((x-0.25)*(x-0.25) + (y-0.25)*(y-0.25) + (z-0.25)*(z-0.25)) * 100.0;

        ijk = (i+Nghost) + (j+Nghost)*jStride + (k+Nghost)*kStride;

        fab[ijk] = 1.0 + std::exp(-r2);

      }
    }
  }

}
